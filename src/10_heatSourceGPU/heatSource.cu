#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include "cuda-samples-master/Common/hip/hip_runtime_api.h"
#include ""
#include "hipblas.h"
#include "hip/device_functions.h"
#include "../common_book/book.h"

#define REAL double
#define MAX_NUMBER 100000
#define ITER_TIME 1000
#define SPEED 0.33f

extern std::vector<std::vector<int>> gAdjInfo; // �洢����ÿ�������ε��ڽ������Σ��ڽ�ָ�������������εİ�Χ���ཻ��
std::vector<std::vector<int>> adjoinTriangles; // �����ε��ڽ�������
extern std::vector<REAL> gIntensity[2];
extern int currentPass; // ���������ʾ��ǰ gIntensity �ǵڼ������� 0 �� 1 ֮�佻��
extern std::vector<int> gSources; // ��Դ��

struct TriangleGPU {
	REAL intensity = 0;
	REAL out = 0;
	int adjInfoSize;
	int adj[20];
};

struct TriangleAdjoin {
	bool isConstant = false;
	REAL intensity = 0;
	int adj[3];
};

//struct TEST {
//	float intensity = 0;
//	// REAL out = 0;
//	int adjInfoSize;
//	//int adj[20];
//};

//__constant__ TEST test[1];

//__constant__ TriangleGPU triangle[40000];
//__device__ TriangleGPU *triangle;

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void heatAdd(TriangleGPU* d, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		d[i].out = d[i].intensity;
		for (int j = 0; j < d[i].adjInfoSize; j++) {
			int tj = d[i].adj[j];
			d[i].out += d[tj].intensity;
		}

		d[i].out /= REAL(d[i].adjInfoSize + 1);
	}
};

__global__ void heatCalculate(TriangleAdjoin* d, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		for (int m = 0; m < ITER_TIME; m++) {
			REAL out = d[i].intensity;
			for (int j = 0; j < 3; j++) {
				int index = d[i].adj[j];
				out += SPEED * (d[index].intensity - d[i].intensity);
			}

			__syncthreads();

			d[i].intensity = out;
			
			__syncthreads();

			if (d[i].isConstant) {
				d[i].intensity = 1;
			}
		}
	}
};

__global__ void heatCalculate2(TriangleAdjoin* d, TriangleAdjoin* o, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		o[i].intensity = d[i].intensity;
		for (int j = 0; j < 3; j++) {
			int index = d[i].adj[j];
			o[i].intensity += SPEED * (d[index].intensity - d[i].intensity);
		}
	}
};

//__global__ void heatAddConstant(int numElements)
//{
//	int i = blockDim.x * blockIdx.x + threadIdx.x;
//
//	if (i < numElements)
//	{
//		REAL out = triangle[i].intensity;
//		for (int j = 0; j < triangle[i].adjInfoSize; j++) {
//			int tj = triangle[i].adj[j];
//			out += triangle[tj].intensity;
//		}
//
//		out /= REAL(triangle[i].adjInfoSize + 1);
//
//		__syncthreads();
//
//		triangle[i].intensity = out;
//	}
//};
//
//extern "C" int readyGPUdata()
//{
//	hipError_t err = hipSuccess;
//	int numElements = gIntensity[currentPass].size();
//
//	size_t size = numElements * sizeof(TriangleGPU);
//	
//	// ׼��GPU����
//	err = hipMalloc((void**)&triangle, size);
//
//	if (err != hipSuccess)
//	{
//		fprintf(stderr, "Failed to allocate device vector d_0 (error code %s)!\n", hipGetErrorString(err));
//		exit(EXIT_FAILURE);
//	}
//
//	// ׼�� CPU ����
//	TriangleGPU* h = (TriangleGPU*)malloc(size);
//
//
//	// Verify that allocations succeeded
//	if (h == NULL)
//	{
//		fprintf(stderr, "Failed to allocate host vectors!\n");
//		exit(EXIT_FAILURE);
//	}
//
//	for (int i = 0; i < numElements; i++) {
//		h[i].intensity = gIntensity[currentPass][i];
//		h[i].adjInfoSize = 0;
//		for (int j = 0; j < 20; j++) {
//			if (j >= gAdjInfo[i].size()) {
//				break;
//			}
//			h[i].adj[j] = gAdjInfo[i][j];
//			h[i].adjInfoSize++;
//		}
//	}
//	
//	//err = hipMemcpyToSymbol(HIP_SYMBOL(triangle), h, size, hipMemcpyHostToDevice);
//	err = hipMemcpy(triangle, h, size, hipMemcpyHostToDevice);
//
//	if (err != hipSuccess)
//	{
//		fprintf(stderr, "Failed to copy vector h from host to device (error code %s)!\n", hipGetErrorString(err));
//		exit(EXIT_FAILURE);
//	}
//
//	return 0;
//}

//extern "C" int doPropogateGPUConstant()
//{
//	// ������ʱ��
//	hipEvent_t start, stop;
//	float elapsedTime;
//	HANDLE_ERROR(hipEventCreate(&start));
//	HANDLE_ERROR(hipEventCreate(&stop));
//	HANDLE_ERROR(hipEventRecord(start, 0));
//
//	// Error code to check return values for CUDA calls
//	hipError_t err = hipSuccess;
//
//	// Print the vector length to be used, and compute its size
//	int numElements = gIntensity[currentPass].size();
//	int prevPass = currentPass;
//	currentPass = 1 - currentPass;
//
//	size_t size = numElements * sizeof(TriangleGPU);
//
//	// ���� GPU ����
//	// Launch the Vector Add CUDA Kernel
//	int threadsPerBlock = 1024;
//	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
//
//	//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
//	heatAddConstant << <blocksPerGrid, threadsPerBlock >> > (numElements);
//
//	err = hipGetLastError();
//
//	if (err != hipSuccess)
//	{
//		fprintf(stderr, "Failed to launch heatAdd kernel (error code %s)!\n", hipGetErrorString(err));
//		exit(EXIT_FAILURE);
//	}
//
//	// �� GPU �Ľ�������� CPU ��
//	// Copy the device result vector in device memory to the host result vector in host memory.
//
//	TriangleGPU* h = (TriangleGPU*)malloc(size);
//	err = hipMemcpy(h, triangle, size, hipMemcpyDeviceToHost);
//	//err = hipMemcpy(h, triangle, size, hipMemcpyDeviceToHost);
//
//	if (err != hipSuccess)
//	{
//		fprintf(stderr, "Failed to copy vector triangle from device to host (error code %s)!\n", hipGetErrorString(err));
//		exit(EXIT_FAILURE);
//	}
//
//	for (int i = 0; i < numElements; i++) {
//		gIntensity[currentPass][i] = h[i].intensity;
//		//printf("the result of %d, %f\n", i, gIntensity[currentPass][i]);
//	}
//	for (int i = 0; i < gSources.size(); i++) {
//		gIntensity[currentPass][gSources[i]] = 1.0;
//	}
//
//	//printf("###################\n");
//
//	// �ͷſռ�
//	// Free device global memory
//	//err = hipFree(d);
//
//	//if (err != hipSuccess)
//	//{
//	//	fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
//	//	exit(EXIT_FAILURE);
//	//}
//
//	//err = hipFree(d_B);
//
//	//if (err != hipSuccess)
//	//{
//	//	fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
//	//	exit(EXIT_FAILURE);
//	//}
//
//	//err = hipFree(d_C);
//
//	//if (err != hipSuccess)
//	//{
//	//	fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
//	//	exit(EXIT_FAILURE);
//	//}
//
//	// Free host memory
//	free(h);
//	//free(h_B);
//	//free(h_C);
//
//	//printf("Done\n");
//
//	// ����ʱ��
//	HANDLE_ERROR(hipEventRecord(stop, 0));
//	HANDLE_ERROR(hipEventSynchronize(stop));
//	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
//	printf("Time taken: %3.1f ms\n", elapsedTime);
//
//	return 0;
//}

extern "C" int doPropogateGPU()
{
	// ������ʱ��
	hipEvent_t start, stop;
	float elapsedTime;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	int numElements = gIntensity[currentPass].size();
	int prevPass = currentPass;
	currentPass = 1 - currentPass;

	size_t size = numElements * sizeof(TriangleGPU);

	// ׼�� CPU ����
	//// Allocate the host input vector h
	//REAL* h = (REAL*)malloc(size);
	TriangleGPU* h = (TriangleGPU *)malloc(size);

	// Verify that allocations succeeded
	if (h == NULL)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < numElements; i++){
		//h[i] = gIntensity[prevPass][i];
		h[i].intensity = gIntensity[prevPass][i];
		h[i].adjInfoSize = 0;
		if (gAdjInfo[i].size() > 20) {
			//printf("right\n");
		}
		for (int j = 0; j < 20; j++) {
			if (j >= gAdjInfo[i].size()) {
				break;
			}
			h[i].adj[j] = gAdjInfo[i][j];
			h[i].adjInfoSize++;
		}
	}

	// ��GPU�ϴ����ڴ�
	// Allocate the device input vector d

	TriangleGPU* d = NULL;
	err = hipMalloc((void**)&d, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_0 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector d_0
	//REAL* d_1 = NULL;
	//err = hipMalloc((void**)&d_1, size);

	//if (err != hipSuccess)
	//{
	//	fprintf(stderr, "Failed to allocate device vector d_1 (error code %s)!\n", hipGetErrorString(err));
	//	exit(EXIT_FAILURE);
	//}

	// �� CPU �ڵ�ֵ������ GPU ��
	// Copy the host input vectors A and B in host memory to the device input vectors in device memory
	//printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// ���� GPU ����
	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 1024;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

	//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	heatAdd << <blocksPerGrid, threadsPerBlock >> > (d, numElements);

	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch heatAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// �� GPU �Ľ�������� CPU ��
	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	//printf("Copy output data from the CUDA device to the host memory\n");
	//err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	free(h);
	h = (TriangleGPU*)malloc(size);
	err = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < numElements; i++) {
		gIntensity[currentPass][i] = h[i].out;
		//printf("the result of %d, %f\n", i, gIntensity[currentPass][i]);
	}
	for (int i = 0; i < gSources.size(); i++) {
		gIntensity[currentPass][gSources[i]] = 1.0;
	}

	//printf("###################\n");

	// �ͷſռ�
	// Free device global memory
	err = hipFree(d);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//err = hipFree(d_B);

	//if (err != hipSuccess)
	//{
	//	fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
	//	exit(EXIT_FAILURE);
	//}

	//err = hipFree(d_C);

	//if (err != hipSuccess)
	//{
	//	fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
	//	exit(EXIT_FAILURE);
	//}

	// Free host memory
	free(h);
	//free(h_B);
	//free(h_C);

	//printf("Done\n");

	// ����ʱ��
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Time taken: %3.1f ms\n", elapsedTime);

	return 0;
}

extern "C" int doPropogateGPU_New()
{
	// ������ʱ��
	hipEvent_t start, stop;
	float elapsedTime;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	int numElements = gIntensity[currentPass].size();
	int prevPass = currentPass;
	currentPass = 1 - currentPass;

	size_t size = numElements * sizeof(TriangleAdjoin);

	// ׼�� CPU ����
	//// Allocate the host input vector h
	//REAL* h = (REAL*)malloc(size);
	TriangleAdjoin* h = (TriangleAdjoin*)malloc(size);

	// Verify that allocations succeeded
	if (h == NULL)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < numElements; i++) {
		//h[i] = gIntensity[prevPass][i];
		h[i].intensity = gIntensity[prevPass][i];
		h[i].isConstant = false;
		for (int j = 0; j < 3; j++) {
			h[i].adj[j] = adjoinTriangles[i][j];
		}
	}
	for (int i = 0; i < gSources.size(); i++) {
		h[gSources[i]].isConstant = true;
	}

	// ��GPU�ϴ����ڴ�
	// Allocate the device input vector d

	TriangleAdjoin* d = NULL;
	err = hipMalloc((void**)&d, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_0 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//TriangleAdjoin* d_o = NULL;
	//err = hipMalloc((void**)&d_o, size);

	//if (err != hipSuccess)
	//{
	//	fprintf(stderr, "Failed to allocate device vector d_0 (error code %s)!\n", hipGetErrorString(err));
	//	exit(EXIT_FAILURE);
	//}

	// Allocate the device input vector d_0
	//REAL* d_1 = NULL;
	//err = hipMalloc((void**)&d_1, size);

	//if (err != hipSuccess)
	//{
	//	fprintf(stderr, "Failed to allocate device vector d_1 (error code %s)!\n", hipGetErrorString(err));
	//	exit(EXIT_FAILURE);
	//}

	// �� CPU �ڵ�ֵ������ GPU ��
	// Copy the host input vectors A and B in host memory to the device input vectors in device memory
	//printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//err = hipMemcpy(d_o, h, size, hipMemcpyHostToDevice);

	//if (err != hipSuccess)
	//{
	//	fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
	//	exit(EXIT_FAILURE);
	//}

	// ���� GPU ����
	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 1024;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

	//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	heatCalculate << <blocksPerGrid, threadsPerBlock >> > (d, numElements);
	//for (int i = 0; i < ITER_TIME; i++) {
		//heatCalculate2 << <blocksPerGrid, threadsPerBlock >> > (d, d_o, numElements);
		//swap(d, d_o);

		//free(h);
		//h = (TriangleAdjoin*)malloc(size);
		//err = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
		//for (int i = 0; i < numElements; i++) {
		//	printf("the result of %d, %f\n", i, h[i].intensity);
		//}
	//}

	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch heatAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// �� GPU �Ľ�������� CPU ��
	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	//printf("Copy output data from the CUDA device to the host memory\n");
	//err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	free(h);
	h = (TriangleAdjoin*)malloc(size);
	err = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < numElements; i++) {
		gIntensity[currentPass][i] = h[i].intensity;
		//printf("the result of %d, %f\n", i, gIntensity[currentPass][i]);
	}
	for (int i = 0; i < gSources.size(); i++) {
		gIntensity[currentPass][gSources[i]] = 1.0;
	}

	//printf("###################\n");

	// �ͷſռ�
	// Free device global memory
	err = hipFree(d);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//err = hipFree(d_B);

	//if (err != hipSuccess)
	//{
	//	fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
	//	exit(EXIT_FAILURE);
	//}

	//err = hipFree(d_C);

	//if (err != hipSuccess)
	//{
	//	fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
	//	exit(EXIT_FAILURE);
	//}

	// Free host memory
	free(h);
	//free(h_B);
	//free(h_C);

	//printf("Done\n");

	// ����ʱ��
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Do %d Times and Time taken: %3.1f ms\n", ITER_TIME, elapsedTime);

	return 0;
}