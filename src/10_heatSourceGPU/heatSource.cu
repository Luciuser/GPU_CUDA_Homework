#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include "cuda-samples-master/Common/hip/hip_runtime_api.h"
#include ""
#include "hipblas.h"
#include "hip/device_functions.h"
#include "../common_book/book.h"

#define REAL double

//extern double *gA, *gB, *gC;
extern int maxLevel;
extern std::vector<std::vector<int>> gAdjInfo; // �洢����ÿ�������ε��ڽ������Σ��ڽ�ָ�������������εİ�Χ���ཻ��
extern std::vector<REAL> gIntensity[2];
extern int currentPass; // ���������ʾ��ǰ gIntensity �ǵڼ������� 0 �� 1 ֮�佻��
extern std::vector<int> gSources; // ��Դ��


struct TriangleGPU {
	REAL intensity = 0;
	REAL out = 0;
	int adjInfoSize;
	int adj[20];
};

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void heatAdd(TriangleGPU* d, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		////C[i] = A[i] + B[i];
		//C[i] = atan(A[i]) / (fabs(sin(fabs(B[i]) + 0.0001)) + 0.1);
		d[i].out = d[i].intensity;
		for (int j = 0; j < d[i].adjInfoSize; j++) {
			int tj = d[i].adj[j];
			d[i].out += d[tj].intensity;
		}

		d[i].out /= REAL(d[i].adjInfoSize + 1);
	}
};

extern "C" int doPropogateGPU()
{
	// ������ʱ��
	hipEvent_t start, stop;
	float elapsedTime;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	int numElements = gIntensity[currentPass].size();
	int prevPass = currentPass;
	currentPass = 1 - currentPass;

	size_t size = numElements * sizeof(TriangleGPU);

	// ׼�� CPU ����
	//// Allocate the host input vector h
	//REAL* h = (REAL*)malloc(size);
	TriangleGPU* h = (TriangleGPU *)malloc(size);

	// Verify that allocations succeeded
	if (h == NULL)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < numElements; i++){
		//h[i] = gIntensity[prevPass][i];
		h[i].intensity = gIntensity[prevPass][i];
		h[i].adjInfoSize = 0;
		if (gAdjInfo[i].size() > 20) {
			//printf("right\n");
		}
		for (int j = 0; j < 20; j++) {
			if (j >= gAdjInfo[i].size()) {
				break;
			}
			h[i].adj[j] = gAdjInfo[i][j];
			h[i].adjInfoSize++;
		}
	}

	// ��GPU�ϴ����ڴ�
	// Allocate the device input vector d

	TriangleGPU* d = NULL;
	err = hipMalloc((void**)&d, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_0 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector d_0
	//REAL* d_1 = NULL;
	//err = hipMalloc((void**)&d_1, size);

	//if (err != hipSuccess)
	//{
	//	fprintf(stderr, "Failed to allocate device vector d_1 (error code %s)!\n", hipGetErrorString(err));
	//	exit(EXIT_FAILURE);
	//}

	// �� CPU �ڵ�ֵ������ GPU ��
	// Copy the host input vectors A and B in host memory to the device input vectors in device memory
	//printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// ���� GPU ����
	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 1024;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

	//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	heatAdd << <blocksPerGrid, threadsPerBlock >> > (d, numElements);

	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch heatAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// �� GPU �Ľ�������� CPU ��
	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	//printf("Copy output data from the CUDA device to the host memory\n");
	//err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	free(h);
	h = (TriangleGPU*)malloc(size);
	err = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < numElements; i++) {
		gIntensity[currentPass][i] = h[i].out;
		//printf("the result of %d, %f\n", i, gIntensity[currentPass][i]);
	}
	for (int i = 0; i < gSources.size(); i++) {
		gIntensity[currentPass][gSources[i]] = 1.0;
	}

	//printf("###################\n");

	// �ͷſռ�
	// Free device global memory
	err = hipFree(d);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	//err = hipFree(d_B);

	//if (err != hipSuccess)
	//{
	//	fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
	//	exit(EXIT_FAILURE);
	//}

	//err = hipFree(d_C);

	//if (err != hipSuccess)
	//{
	//	fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
	//	exit(EXIT_FAILURE);
	//}

	// Free host memory
	free(h);
	//free(h_B);
	//free(h_C);

	//printf("Done\n");

	// ����ʱ��
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Time taken: %3.1f ms\n", elapsedTime);

	return 0;
}
